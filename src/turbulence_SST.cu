#include "hip/hip_runtime.h"
#include"turbulence_SST.cuh"
#include "common.h"
#include "Global_var.h"
#include "Flow_var.h"
#include "sub_turbulence_SST.h"
#include "cmath"

__device__
double cuda_min(double a, double b) {
	return a < b ? a : b;
}

__device__
double cuda_max(double a, double b) {
	return a > b ? a : b;
}


void Amut_boundary(int mBlock);

__global__
void cuda_calcu_Qt(int *transferInt, double * transferDouble, double * x1, double *y1, double * d, double * uu, double * vv, double *Kt, double * Wt,
	double *dw, double *x, double * y, double *f1, double *Amu, double *Amu_t, double * Res5, double * Res6);

__global__
void cuda_calcu_i_devdiff(int *transferInt, double * transferDouble, double * x, double * y, double *uu, double * vv,
	double *x1, double * y1, double * Kt, double * Wt, double *f1, double * Amu, double * Amu_t, double *Res5, double * Res6);

__global__
void cuda_calcu_j_devdiff(int *transferInt, double * transferDouble, double * x, double * y, double *uu, double * vv,
	double *x1, double * y1, double * Kt, double * Wt, double *f1, double * Amu, double * Amu_t, double *Res5, double *Res6);


void turbulence_SST_kw_before_cuda(double * Amu_t_dev, double * Amu_dev, double *d, double *uu, double *vv, double * T,
	double *U_dev,double *x_dev, double *y_dev, double *x1_dev, double *y1_dev,int mBlock, flow_var & fl, int * transferInt_dev, double * transferDouble_dev)
{
	Block_TYPE & B = Mesh[1].Block[mBlock];
	const int nx =  B.nx;	const int ny =  B.ny;
	const int mm1 = nx + 2 * LAP - 1;	const int nn1 = ny + 2 * LAP - 1;
	const int mm = nx + 2 * LAP;	const int nn = ny + 2 * LAP;

	dim3 threadPerBlock(16, 16);
	dim3  blockPerGrid((nx + 2 * LAP + 1 + threadPerBlock.x - 1) / threadPerBlock.x, (ny + 2 * LAP + 1 + threadPerBlock.y - 1) / threadPerBlock.y);

	//�����������Ϣ�������豸
	double * dw_dev;	double *dw_host;
	HANDLE_ERROR(hipHostMalloc((double **)& dw_host, (nx + 1)*(ny + 1) * sizeof(double)));
	HANDLE_ERROR(hipMalloc((double **)&dw_dev, (nx + 1)*(ny + 1) * sizeof(double)));
	for (int i = 1; i <= nx; ++i) {
		for (int j = 1; j <= ny; ++j) {
			dw_host[i*(ny + 1) + j] =  B.dw[i][j];
		}
	}
	HANDLE_ERROR(hipMemcpy(dw_dev, dw_host, (nx + 1)*(ny + 1) * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipHostFree(dw_host));


	//������ճ��ϵ������ֵ
	HANDLE_ERROR(hipMemset(Amu_t_dev, 0, (mm1 + 1)*(nn1 + 1) * sizeof(double)));
	
	/*return*/
	return;


	double * Kt; double *  Wt; double * f1;		//[nx+1][ny+1]
	//double * Qk; double * Qw;		// [mm1 + 1][nn1 + 1]
	
	HANDLE_ERROR(hipMalloc((double **)& f1, (nx + 1)*(ny + 1) * sizeof(double)));
	HANDLE_ERROR(hipMemset(f1, 0, (nx + 1)*(ny + 1) * sizeof(double)));	//���㺯��f1 �����ֽ�������Զ������

	//HANDLE_ERROR(hipMalloc((double **)& Qk, (nx + 1)*(ny + 1) * sizeof(double)));
	//HANDLE_ERROR(hipMalloc((double **)& Qw, (nx + 1)*(ny + 1) * sizeof(double)));

	HANDLE_ERROR(hipMalloc((double **)& Kt, (mm1 + 1)*(nn1 + 1) * sizeof(double)));
	HANDLE_ERROR(hipMalloc((double **)& Wt, (mm1 + 1)*(nn1 + 1) * sizeof(double)));

	double * Res5_dev;	double * Res6_dev;	//[nx+1][ny+1];
	HANDLE_ERROR(hipMalloc((double **)& Res5_dev, (nx + 1)*(ny + 1) * sizeof(double)));
	HANDLE_ERROR(hipMalloc((double **)& Res6_dev, (nx + 1)*(ny + 1) * sizeof(double)));

	//����ҳ�����ڴ潫ʣ�������������豸
	double *Kt_host; double * Wt_host;
	HANDLE_ERROR(hipHostAlloc((double **)&Kt_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((double **)&Wt_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipHostMallocDefault));
	for (int i = 1; i <= mm1; ++i) {
		for (int j = 1; j <= nn1; ++j) {
			int flag = i*(nn1 + 1) + j;
			Kt_host[flag] =  B.U[i][j][5] / fl.d[i][j];
			Wt_host[flag] =  B.U[i][j][6] / fl.d[i][j];
		}
	}
	HANDLE_ERROR(hipMemcpy(Kt, Kt_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(Wt, Wt_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipHostFree(Kt_host));
	HANDLE_ERROR(hipHostFree(Wt_host));

	//��Դ��
	cuda_calcu_Qt<<<blockPerGrid, threadPerBlock>>>(transferInt_dev, transferDouble_dev, x1_dev, y1_dev, d, uu, vv, Kt, Wt, dw_dev,
													x_dev, y_dev, f1, Amu_dev, Amu_t_dev,  Res5_dev, Res6_dev);

	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGetLastError());

	//����ɢ��Ͷ�����
	cuda_calcu_i_devdiff << <blockPerGrid, threadPerBlock >> >(transferInt_dev, transferDouble_dev, x_dev, y_dev, uu, vv,
		x1_dev, y1_dev, Kt, Wt, f1, Amu_dev, Amu_t_dev, Res5_dev, Res6_dev);

	cuda_calcu_j_devdiff << <blockPerGrid, threadPerBlock >> >(transferInt_dev, transferDouble_dev, x_dev, y_dev, uu, vv,
		x1_dev, y1_dev, Kt, Wt, f1, Amu_dev, Amu_t_dev, Res5_dev, Res6_dev);

	//���������ݿ�����������
	double *Res5_host;	double *Res6_host;
	HANDLE_ERROR(hipHostAlloc((double **) & Res5_host, (nx + 1)*(ny + 1) * sizeof(double), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((double **)& Res6_host, (nx + 1)*(ny + 1) * sizeof(double), hipHostMallocDefault));
	HANDLE_ERROR(hipMemcpy(Res5_host, Res5_dev, (nx + 1)*(ny + 1) * sizeof(double), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(Res6_host, Res6_dev, (nx + 1)*(ny + 1) * sizeof(double), hipMemcpyDeviceToHost));

	double *Amu_host;	double *Amu_t_host;
	HANDLE_ERROR(hipHostAlloc((double **)& Amu_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((double **)& Amu_t_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipHostMallocDefault));
	HANDLE_ERROR(hipMemcpy(Amu_host, Amu_dev, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(Amu_t_host, Amu_t_dev, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyDeviceToHost));
	for (int i = 0; i < (mm1 + 1)*(nn1 + 1); ++i) {
		Amu_host[i] = Amu_host[i] / Re;
		Amu_t_host[i] = Amu_t_host[i] / Re;
	}
	HANDLE_ERROR(hipMemcpy(Amu_dev, Amu_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(Amu_t_dev, Amu_t_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));

	for (int i = 1; i <= nx - 1; ++i) {
		for (int j = 1; j <= ny - 1; ++j) {
			int flag1 = i*(ny + 1) + j;
			B.Res[i][j][5] = Res5_host[flag1];
			B.Res[i][j][6] = Res6_host[flag1];
			int myi = i + LAP;	int myj = j + LAP;
			int flag2 = myi*(nn1 + 1) + myj;
			B.Amu[myi][myj] = Amu_host[flag2];
			B.Amu_t[myi][myj] = Amu_t_host[flag2];

			/*printf("%d, %d\n", i, j);
			printf("%e, %e, %e, %e\n", B.Res[i][j][5], B.Res[i][j][6], B.Amu[myi][myj], B.Amu_t[myi][myj]);
*/
			/*if (i == 3 && j == 14) {
				printf("T= %e, Amu= %e\n", T[flag], Amu[flag]);
			}*/
		}
	}
	//PAUSE;

	HANDLE_ERROR(hipHostFree(Res5_host));
	HANDLE_ERROR(hipHostFree(Res6_host));
	HANDLE_ERROR(hipHostFree(Amu_host));
	HANDLE_ERROR(hipHostFree(Amu_t_host));


	//�趨����ճ��ϵ���������ֵ
	Amut_boundary(mBlock);

	HANDLE_ERROR(hipFree(f1));
	HANDLE_ERROR(hipFree(dw_dev));
	//HANDLE_ERROR(hipFree(Qk));	HANDLE_ERROR(hipFree(Qw));
	HANDLE_ERROR(hipFree(Kt)); HANDLE_ERROR(hipFree(Wt));
	HANDLE_ERROR(hipFree(Res5_dev)); HANDLE_ERROR(hipFree(Res6_dev));
}


__global__
void cuda_calcu_Qt(int *transferInt, double * transferDouble, double * x1, double *y1,double * d, double * uu, double * vv,	double *Kt, double * Wt, 
	double *dw, double *x, double * y, double *f1, double *Amu, double *Amu_t,  double * Res5, double * Res6)
{
	__shared__  int nx, ny, LAP, nn1, nn;
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	nx = transferInt[0];	ny = transferInt[1];
	LAP = transferInt[2];
	nn = ny + 2 * LAP;
	nn1 = ny + 2 * LAP-1;

	double Re = transferDouble[5];

	if (i >= 1 && i <= nx - 1 && j >= 1 && j <= ny - 1) {
		int myi = i + LAP;	int myj = j + LAP;

		//ģ��ϵ��(1Ϊk - wģ�͵�ϵ�����ڽ�����ʹ�ã� 2Ϊk - epslģ�͵�ϵ����Զ����ʹ�ã�
		//SST��һ��k - w��k - epslģ�͵Ļ��ģ�ͣ�ͨ�����غ���f���л�
		const double  beta1_SST = 0.075E0, Cw1_SST = 0.533E0;
		const double  sigma_w2_SST = 0.856E0, beta2_SST = 0.0828E0, Cw2_SST = 0.440E0;
		const double a1_SST = 0.31E0, betas_SST = 0.09E0;

		double Dix, Diy, Djx, Djy, Ds, Dik, Diw, Djk, Djw, Diu, Div, Dju, Djv, kx, ky, Wx, Wy, ux, vx, uy, vy;
		double  omega, Kws, CD_kw, arg1, arg2, arg3, f2, t11, t22, t12, Pk, Pk0;

		int flag = i*(ny + 1) + j;
		int flag1 = (myi + 1)*(nn1 + 1) + myj;
		int flag2 = (myi - 1)*(nn1 + 1) + myj;
		int flag3 = myi*(nn1 + 1) + myj;

		//����������������x, y�ĵ�����ʹ��Jocabian�任
		Dix = ( x1[flag1] -  x1[flag2])*0.5E0;
		Diy = ( y1[flag1] -  y1[flag2])*0.5E0;
		Djx = ( x1[flag3 + 1] -  x1[flag3 - 1])*0.5E0;
		Djy = ( y1[flag3 + 1] -  y1[flag3 - 1])*0.5E0;
		Ds = 1.E0 / (Dix*Djy - Djx*Diy);

		Diu = (uu[flag1] - uu[flag2])*0.5E0;
		Div = (vv[flag1] - vv[flag2])*0.5E0;
		Dik = (Kt[flag1] - Kt[flag2])*0.5E0;
		Diw = (Wt[flag1] - Wt[flag2])*0.5E0;

		Dju = (uu[flag3 + 1] - uu[flag3 - 1])*0.5E0;
		Djv = (vv[flag3 + 1] - vv[flag3 - 1])*0.5E0;
		Djk = (Kt[flag3 + 1] - Kt[flag3 - 1])*0.5E0;
		Djw = (Wt[flag3 + 1] - Wt[flag3 - 1])*0.5E0;

		//����ֵ
		ux = (Diu*Djy - Dju*Diy)*Ds;
		vx = (Div*Djy - Djv*Diy)*Ds;
		kx = (Dik*Djy - Djk*Diy)*Ds;
		Wx = (Diw*Djy - Djw*Diy)*Ds;

		uy = (-Diu*Djx + Dju*Dix)*Ds;
		vy = (-Div*Djx + Djv*Dix)*Ds;
		ky = (-Dik*Djx + Djk*Dix)*Ds;
		Wy = (-Diw*Djx + Djw*Dix)*Ds;

		//��������ճ��ϵ�� �� Blazek's Book Eq. (7.66)    
		//if (i == 3 && j == 14) { printf("AMu= %e  ", Amu[flag3]); }
		Amu[flag3] = Amu[flag3] * Re;
		/*if (i == 3 && j == 14) {
			printf("   %e\n", Amu[flag3]);
		}*/

		omega = vx - uy;      //����
		arg2 = cuda_max(2.E0* sqrt(abs(Kt[flag3])) / (0.09*Wt[flag3] * dw[flag] * Re), 500.E0*Amu[flag3] / 
													(d[flag3] * Wt[flag3] * dw[flag] * dw[flag] * Re*Re));

		f2 = tanh(arg2*arg2);

		//////Revised by Wang XiangYu
		Amu_t[flag3] = cuda_min(cuda_min(d[flag3] * Kt[flag3] / Wt[flag3], a1_SST*d[flag3] * Kt[flag3] * Re / (f2*abs(omega))), 100000.);
		//����f1(ʶ���Ƿ�Ϊ��������������������1��
		Kws = 2.E0*(kx*Wx + ky*Wy)*d[flag3] * sigma_w2_SST / (Wt[flag3] + 1.E-20);      //����������
		CD_kw = cuda_max(Kws, 1.E-20);
		arg3 = cuda_max(sqrt(abs(Kt[flag3])) / (0.09*Wt[flag3] * dw[flag] * Re), 500.E0*Amu[flag3] / (d[flag3] * Wt[flag3] * dw[flag] * dw[flag] * Re*Re));

		arg1 = cuda_min(arg3, 4.E0*d[flag3] * sigma_w2_SST*Kt[flag3] / (CD_kw*dw[flag] * dw[flag]));
		f1[flag] = tanh(arg1*arg1*arg1*arg1);             //���غ�����������������1��Զ����������0  �������л�k - w��k - epsl����)

														  //��Ӧ�� ��ʹ������ճģ�ͣ�
		t11 = ((4.E0 / 3.E0)*ux - (2.E0 / 3.E0)*vy)*Amu_t[flag3] - (2.E0 / 3.E0)*d[flag3] * Kt[flag3] * Re;   //Blazek's Book, Eq. (7.25)
		t22 = ((4.E0 / 3.E0)*vy - (2.E0 / 3.E0)*ux)*Amu_t[flag3] - (2.E0 / 3.E0)*d[flag3] * Kt[flag3] * Re;
		t12 = (uy + vx)*Amu_t[flag3];

		//���ܷ��̵�Դ����� - ��ɢ)
		//Pk = t11*ux + t22*vy + t12*(uy + vx)  //���������� ����Ӧ������Ӧ���ʣ�
		Pk = Amu_t[flag3] * omega*omega;        //��

		Pk0 = cuda_min(Pk, 20.E0*betas_SST*Kt[flag3] * Wt[flag3] * Re*Re);    //������������������ƣ���ֹ���ܹ���

		double Qk = Pk0 / Re - betas_SST*d[flag3] * Wt[flag3] * Kt[flag3] * Re;    //k���̵�Դ��  �������� - ��ɢ�

		double Cw_SST = f1[flag] * Cw1_SST + (1.E0 - f1[flag])*Cw2_SST;    //ģ��ϵ��������f1���������л�
		double beta_SST = f1[flag] * beta1_SST + (1.E0 - f1[flag])*beta2_SST;    //ģ��ϵ��������f1���������л�
		//Qw[flag] = Cw_SST*d[flag]*Pk / ( Amu_t[flag] + 1.d - 20) / Re - beta_SST*d[flag]*Wt[flag]**2 * Re + (1.E0 - f1[flag])*Kws / Re     //W���̵�Դ��
		double Qw = Cw_SST*d[flag3] * omega*omega / Re - beta_SST*d[flag3] * Wt[flag3] * Wt[flag3] * Re + (1.E0 - f1[flag])*Kws / Re;     //W���̵�Դ��

		//����õ�Ԫ����� 
		flag1 = (i + LAP)*(nn + 1) + j + LAP;	flag2 = (i + 1 + LAP)*(nn + 1) + j + LAP;
		const double vol = abs((x[flag1] - x[flag2 + 1])*(y[flag2] - y[flag1 + 1]) -
			(x[flag2] - x[flag1 + 1])*(y[flag1] - y[flag2 + 1]))*0.5e0;
		
		Res5[flag] += Qk * vol;
		Res6[flag] += Qw * vol;
	}
}


//������������ɢ��
__global__
void cuda_calcu_i_devdiff(int *transferInt, double * transferDouble, double * x, double * y, double *uu, double * vv,
	double *x1, double * y1, double * Kt, double * Wt, double *f1, double * Amu, double * Amu_t, double *Res5, double * Res6)
{
	__shared__  int nx, ny, LAP, nn,  nn1;
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	nx = transferInt[0];	ny = transferInt[1];
	LAP = transferInt[2];
	nn = ny + 2 * LAP;
	nn1 = ny + 2 * LAP - 1;
	double Re = transferDouble[5];

	if (i >= 1 && i <= nx && j >= 1 && j <= ny - 1) {

		const double sigma_k1_SST = 0.85E0, sigma_w1_SST = 0.5E0;
		const double sigma_k2_SST = 1.E0, sigma_w2_SST = 0.856E0;
		double sigma_K_SST, sigma_W_SST;
		double Dix, Diy, Djx, Djy, Ds, Dik, Diw, Djk, Djw, Diu, Div, Dju, Djv, kx, ky, Wx, Wy;

		int myi = i + LAP;	int myj = j + LAP;
		int flagL = (i + LAP)*(nn + 1) + j + 1 + LAP;
		int flagR = (i + LAP)*(nn + 1) + j + LAP;
		double dx = x[flagL] - x[flagR];
		double dy = y[flagL] - y[flagR];
		const double si = sqrt(dx*dx + dy*dy);	//�߳�
		const double ni1 = dy / si;
		const double ni2 = -dx / si;   //normal vector at(i, j) or (I - 1 / 2, J)

		int flag = i*(ny + 1) + j;
		int flag1 = myi*(nn1 + 1) + myj;
		int flag2 = (myi - 1)*(nn1 + 1) + myj;
									   //���������1��ӭ���ʽ
		double un1 = uu[flag2] * ni1 + vv[flag2] * ni2;
		double un2 = uu[flag1] * ni1 + vv[flag1] * ni2;

		//1��L - F ��ʽ
		double Fluxk= -0.5E0*((un1 + abs(un1))*Kt[flag2] + (un2 - abs(un2))*Kt[flag1])*si;
		double Fluxw = -0.5E0*((un1 + abs(un1))*Wt[flag2] + (un2 - abs(un2))*Wt[flag1])*si;

		__syncthreads();

		//ճ�����ɢ�������2�����ĸ�ʽ

		//��ʽϵ����k - w��k - epsl��ʽϵ��֮��ѡ��(f1��Ϊ�л����غ���)
		sigma_K_SST = f1[flag] * sigma_k1_SST + (1.E0 - f1[flag])*sigma_k2_SST;
		sigma_W_SST = f1[flag] * sigma_w1_SST + (1.E0 - f1[flag])*sigma_w2_SST;

		//�����ϵ�ֵ = ����ֵ��ƽ��, �߽��ϵ���ɢϵ�� = �ڲ��ֵ
		double Amu1, Amu2;
		if (i == 1) {
			Amu1 = Amu[flag1] + sigma_K_SST*Amu_t[flag1];         //��ɢϵ��(k����)
			Amu2 = Amu[flag1] + sigma_W_SST*Amu_t[flag1];         //��ɢϵ��(w����)
		}
		else if (i == nx) {
			Amu1 = Amu[flag2] + sigma_K_SST*Amu_t[flag2];        //��ɢϵ��(k����)
			Amu2 = Amu[flag2] + sigma_W_SST*Amu_t[flag2];        //��ɢϵ��(w����)
		}
		else {
			Amu1 = (Amu[flag2] + Amu[flag1] + sigma_K_SST*(Amu_t[flag2] + Amu_t[flag1]))*0.5E0;        //��ɢϵ��(k����), �����ϵ�ֵ = ����ֵ��ƽ��
			Amu2 = (Amu[flag2] + Amu[flag1] + sigma_W_SST*(Amu_t[flag2] + Amu_t[flag1]))*0.5E0;        //��ɢϵ��(w����)
		}

		//������������k, w��������x, y�ĵ��� ������Jocabian�任��
		//----Jocabianϵ�� ����������Լ�������ĵ���, ���ڼ����������ĵ�����
		Dix = x1[flag1] - x1[flag2];
		Diy = y1[flag1] - y1[flag2];
		Djx = (x1[flag2 + 1] + x1[flag1 + 1] - x1[flag2 - 1] - x1[flag1 - 1])*0.25E0;
		Djy = (y1[flag2 + 1] + y1[flag1 + 1] - y1[flag2 - 1] - y1[flag1 - 1])*0.25E0;
		Ds = 1.E0 / (Dix*Djy - Djx*Diy);
		//�������Լ�������ĵ���
		Dik = Kt[flag1] - Kt[flag2];
		Diw = Wt[flag1] - Wt[flag2];
		Djk = (Kt[flag2 + 1] + Kt[flag1 + 1] - Kt[flag2 - 1] - Kt[flag1 - 1])*0.25E0;
		Djw = (Wt[flag2 + 1] + Wt[flag1 + 1] - Wt[flag2 - 1] - Wt[flag1 - 1])*0.25E0;
		//��������x, y����ĵ���
		kx = (Dik*Djy - Djk*Diy)*Ds;
		Wx = (Diw*Djy - Djw*Diy)*Ds;
		ky = (-Dik*Djx + Djk*Dix)*Ds;
		Wy = (-Diw*Djx + Djw*Dix)*Ds;
		//ճ��Ӧ��������ͨ��
		Fluxk += Amu1*(kx*ni1 + ky*ni2)*si / Re;
		Fluxw += Amu2*(Wx*ni1 + Wy*ni2)*si / Re;

		Res5[flag] -= Fluxk;
		Res6[flag] -= Fluxw;

		__syncthreads();
		flag = (i - 1)*(ny + 1) + j;
		Res5[flag] += Fluxk;
		Res6[flag] += Fluxw;
	}
}


//������������ɢ��
__global__
void cuda_calcu_j_devdiff(int *transferInt, double * transferDouble, double * x, double * y, double *uu, double * vv,
	double *x1, double * y1, double * Kt, double * Wt, double *f1, double * Amu, double * Amu_t, double *Res5, double *Res6)
{
	__shared__  int nx, ny, LAP, nn, nn1;
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	nx = transferInt[0];	ny = transferInt[1];
	LAP = transferInt[2];
	nn = ny + 2 * LAP;
	nn1 = ny + 2 * LAP - 1;
	double Re = transferDouble[5];

	if (i >= 1 && i <= nx-1 && j >= 1 && j <= ny) {

		const double sigma_k1_SST = 0.85E0, sigma_w1_SST = 0.5E0;
		const double sigma_k2_SST = 1.E0, sigma_w2_SST = 0.856E0;
		double sigma_K_SST, sigma_W_SST;
		double Dix, Diy, Djx, Djy, Ds, Dik, Diw, Djk, Djw, Diu, Div, Dju, Djv, kx, ky, Wx, Wy;
	
		//�߳���������
		int flagL = (i + 1 + LAP)*(nn + 1) + j + LAP;
		int flagR = (i + LAP)*(nn + 1) + j + LAP;
		double dx = x[flagL] - x[flagR];
		double dy = y[flagL] - y[flagR];
		const double sj = sqrt(dx*dx + dy*dy);	//�߳�
		const double nj1 = -dy / sj;
		const double nj2 = dx / sj;   //normal vector at(i, j) or (I - 1 / 2, J)

									  //���������1��ӭ���ʽ ��L - F���ѣ�
		int myi = i + LAP;	int myj = j + LAP;
		int flag = i*(ny + 1) + j;
		int flag1 = myi*(nn1 + 1) + myj;
		int flag2 = (myi + 1)*(nn1 + 1) + myj;
		int flag3 = (myi - 1)*(nn1 + 1) + myj;

		double un1 =  uu[flag1 - 1] * nj1 + vv[flag1 - 1] * nj2;
		double un2 =  uu[flag1] * nj1 + vv[flag1] * nj2;
		double Fluxk= -0.5E0*((un1 + abs(un1))*Kt[flag1 - 1] + (un2 - abs(un2))*Kt[flag1])*sj;
		double Fluxw= -0.5E0*((un1 + abs(un1))*Wt[flag1 - 1] + (un2 - abs(un2))*Wt[flag1])*sj;

		__syncthreads();
		//ճ����
		//-------- - Vmyiscous term---------------------------------------------------------------------------- -
		sigma_K_SST = f1[flag] * sigma_k1_SST + (1.E0 - f1[flag])*sigma_k2_SST;
		sigma_W_SST = f1[flag] * sigma_w1_SST + (1.E0 - f1[flag])*sigma_w2_SST;
		double Amu1, Amu2;
		if (j == 1) {
			Amu1 =  Amu[flag1] + sigma_K_SST* Amu_t[flag1];
			Amu2 =  Amu[flag1] + sigma_W_SST* Amu_t[flag1];
		}
		else if (j == ny) {
			Amu1 =  Amu[flag1 - 1] + sigma_K_SST* Amu_t[flag1 - 1];
			Amu2 =  Amu[flag1 - 1] + sigma_W_SST* Amu_t[flag1 - 1];
		}
		else {
			Amu1 = ( Amu[flag1] +  Amu[flag1 - 1] + sigma_K_SST*( Amu_t[flag1] +  Amu_t[flag1 - 1]))*0.5E0;
			Amu2 = ( Amu[flag1] +  Amu[flag1 - 1] + sigma_W_SST*( Amu_t[flag1] +  Amu_t[flag1 - 1]))*0.5E0;
		}

		//������������k, w��������x, y�ĵ��� ������myjocabmyian�任��
		Dix = ( x1[flag2 - 1] +  x1[flag2] -  x1[flag3 - 1] -  x1[flag3])*0.25E0;
		Diy = ( y1[flag2 - 1] +  y1[flag2] -  y1[flag3 - 1] -  y1[flag3])*0.25E0;
		Djx =  x1[flag1] -  x1[flag1 - 1];
		Djy =  y1[flag1] -  y1[flag1 - 1];
		Ds = 1.E0 / (Dix*Djy - Djx*Diy);

		Dik = (Kt[flag2 - 1] + Kt[flag2] - Kt[flag3 - 1] - Kt[flag3])*0.25E0;
		Diw = (Wt[flag2 - 1] + Wt[flag2] - Wt[flag3 - 1] - Wt[flag3])*0.25E0;
		Djk = Kt[flag1] - Kt[flag1 - 1];
		Djw = Wt[flag1] - Wt[flag1 - 1];
		//
		kx = (Dik*Djy - Djk*Diy)*Ds;
		Wx = (Diw*Djy - Djw*Diy)*Ds;
		ky = (-Dik*Djx + Djk*Dix)*Ds;
		Wy = (-Diw*Djx + Djw*Dix)*Ds;

		Fluxk += Amu1*(kx*nj1 + ky*nj2)*sj / Re;
		Fluxw += Amu2*(Wx*nj1 + Wy*nj2)*sj / Re;
		__syncthreads();
	
		Res5[flag] -= Fluxk;
		Res6[flag] -= Fluxw;

		__syncthreads();
		flag = i*(ny + 1) + j - 1;
		Res5[flag] += Fluxk;
		Res6[flag] += Fluxw;
	}
}


//ճ��ϵ���������ϵ�ֵ ���̱ڱ߽���÷�ֵ���Ա�֤�̱��ϵ�ƽ������ճ��ϵ��Ϊ0��
void Amut_boundary(int mBlock)
{
	int ib, ie, jb, je;
	Block_TYPE & B = Mesh[1].Block[mBlock];

	int nx = B.nx; int ny = B.ny;

	//Ghost Cell ��� mutֵΪ �ڵ�mutֵ*�� - 1��(��������ʹ������mut = 0)

	for (int ksub = 1; ksub <= B.subface; ++ksub) {
		BC_MSG_TYPE & Bc = B.bc_msg[ksub];
		if (Bc.neighb == BC_Wall) {   //(ճ��)����߽�����
			ib = Bc.ist; ie = Bc.iend;
			jb = Bc.jst; je = Bc.jend;

			if (Bc.face == 1) {   //i -
				for (int j = jb; j <= je - 1; ++j) {
					B.Amu_t[0 + LAP][j + LAP] = -B.Amu_t[1 + LAP][j + LAP];
				}
			}
			else if (Bc.face == 3) {   //i +
				for (int j = jb; j <= je - 1; ++j) {
					B.Amu_t[nx + LAP][j + LAP] = -B.Amu_t[nx - 1 + LAP][j + LAP];      //mut
				}
			}
			else if (Bc.face == 2) {   //j -
				for (int i = ib; i <= ie - 1; ++i) {
					B.Amu_t[i + LAP][0 + LAP] = -B.Amu_t[i + LAP][1 + LAP];       //mut
				}
			}
			else if (Bc.face == 4) {   //j +
				for (int i = ib; i <= ie - 1; ++i) {
					B.Amu_t[i + LAP][ny + LAP] = -B.Amu_t[i + LAP][ny - 1 + LAP];       //mut
				}
			}
		}
	}
}

