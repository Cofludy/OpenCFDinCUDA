#include "hip/hip_runtime.h"
#include "kernelResidual.h"
#include "sub_Residual.h"
#include "common.h"
#include "Global_var.h"
#include "Flow_var.h"
#include"turbulence_SST.cuh"
#include<cstring>
#include <cmath>

__global__
void getResidual_i_WithKernel(int * transferInt, double *transferDouble, double* U, double * Amu, double *Amu_t,
	double * x1, double *y1, double *x, double *y, double *Fluxi, double *uu, double *vv, double * T);

__global__
void getResidual_j_WithKernel(int * transferInt, double *transferDouble, double* U, double * Amu, double *Amu_t,
	double * x1, double *y1, double *x, double *y, double *Fluxj, double *uu, double *vv, double * T);

__global__
void comput_duvtpckw_with_cuda(int * transferInt, double *transferDouble,  double * d,
	double * uu, double * vv, double *T, double * U);


__global__
void get_viscous_with_cuda(int * transferInt, double *transferDouble, double *Amu, double * T);

__device__
void cuda_scheme_fm(double & UR, double u1, double u2, double u3, double u4);

__device__
void cuda_scheme_fP(double &UL, double u1, double u2, double u3, double u4);

__device__
void cuda_Reconstuction_Characteristic(double U0[5][5], double *UL, double * UR, double gamma);

__device__
void  cuda_Flux_Van_Leer_1Da(double *QL, double * QR, double *Flux, const double gamma);

__device__
double  minmod(double a, double b)
{
	double  minmod;
	if (a*b < 0) {
		minmod = 0;
	}
	else {
		minmod = abs(a) <= abs(b) ? a : b;
	}
	return minmod;
}

__global__
void checkNum(int hi, int hj, double * vector, int colum)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;

	if (i == hi && j == hj) {
		printf("Gpu= %d, %d, %f\n", i, j, vector[i*colum + j]);
	}
}

__global__
void checkNum(int hi, int hj,int k, double * vector, int colum, int zco)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;

	if (i == hi && j == hj) {
		printf("GPU=  %d, %d, %f\n", i, j, vector[i*colum*zco + j*zco+k]);
	}
}

void beforeKernelResidual(int mBlock, flow_var  & fl)
{
	HANDLE_ERROR(hipSetDevice(0));

	Block_TYPE & B = Mesh[1].Block[mBlock];
	const int nx = B.nx;	const int ny = B.ny;
	
	dim3 threadPerBlock(16, 16);
	dim3  blockPerGrid((nx + 2 * LAP + 1 + threadPerBlock.x - 1) / threadPerBlock.x, (ny + 2 * LAP + 1 + threadPerBlock.y - 1) / threadPerBlock.y);

	//���غ�������ݵ�GPU��
	const int mm1 = nx + 2 * LAP - 1;	const int nn1 = ny + 2 * LAP - 1;
	double * U_dev;	
	HANDLE_ERROR(hipMalloc((double **)& U_dev, (mm1 + 1)*(nn1 + 1) * 5 * sizeof(double)));
	double * U_host;	//�����ϵ�ҳ�����ڴ�
	HANDLE_ERROR(hipHostAlloc((double **)&U_host, (mm1 + 1)*(nn1 + 1) * 5 * sizeof(double), hipHostMallocDefault));
	for (int i = 1; i <= mm1; ++i) {
		for (int j = 1; j <= nn1; ++j) {
				memcpy(U_host + i*(nn1 + 1) * 5 + j * 5, B.U[i][j], 5 * sizeof(double));
		}
	}

	HANDLE_ERROR(hipMemcpy(U_dev, U_host, (mm1 + 1)*(nn1 + 1) * 5 * sizeof(double), hipMemcpyHostToDevice));

	//for (int i = 1; i <= mm1; ++i) {
	//	for (int j = 1; j <= nn1; ++j) {
	//		for (int k = 1; k <= 4; ++k) {
	//			checkNum << <blockPerGrid, threadPerBlock >> > (i, j, k, U_dev, nn1 + 1, 5);
	//			printf("CPU Host =  %f ",U_host[i*(nn1+1)*5+j*5+k]);
	//			printf("CPU = %d, %d, %f ",i, j, B.U[i][j][k]);
	//			hipDeviceSynchronize();
	//		}
	//	}
	//}

	HANDLE_ERROR(hipHostFree(U_host));		U_host = nullptr;

	//���������
	double *d, double *uu; double *vv; double *T; 
	HANDLE_ERROR(hipMalloc((double **)& d, (mm1 + 1)*(nn1 + 1) * sizeof(double)));
	HANDLE_ERROR(hipMalloc((double **)& uu, (mm1 + 1)*(nn1 + 1) * sizeof(double)));
	HANDLE_ERROR(hipMalloc((double **)& vv, (mm1 + 1)*(nn1 + 1) * sizeof(double)));
	HANDLE_ERROR(hipMalloc((double **)& T, (mm1 + 1)*(nn1 + 1) * sizeof(double)));

	comput_duvtpckw_with_cuda<<<blockPerGrid, threadPerBlock>>> (transferInt_dev, transferDouble_dev, d, uu, vv, T, U_dev);	//����kernel �������������

	//{//���ﲻ���豸�˼�������������������˼���������������ݿ�����������
	//	double * temp_host;
	//	HANDLE_ERROR(hipHostAlloc((double **)& temp_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipHostMallocDefault));
	//	const int colum_size = nn1 + 1;
	//	const int colum_size_bite = (nn1+1) * sizeof(double);
	//	//copy d
	//	for (int i = 0; i <= mm1; ++i) {
	//		memcpy( temp_host + i*colum_size, fl.d[i], colum_size_bite);
	//	}
	//	HANDLE_ERROR(hipMemcpy(d, temp_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));
	//	//copy uu
	//	for (int i = 0; i <= mm1; ++i) {
	//		memcpy(temp_host + i*colum_size, fl.uu[i], colum_size_bite);
	//	}
	//	HANDLE_ERROR(hipMemcpy(uu, temp_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));
	//	//copyt vv
	//	for (int i = 0; i <= mm1; ++i) {
	//		memcpy(temp_host + i*colum_size, fl.v[i], colum_size_bite);
	//	}
	//	HANDLE_ERROR(hipMemcpy(vv, temp_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));
	//	//copy T
	//	for (int i = 0; i <= mm1; ++i) {
	//		memcpy(temp_host + i*colum_size, fl.T[i], colum_size_bite);
	//	}
	//	HANDLE_ERROR(hipMemcpy(T, temp_host, (mm1 + 1)*(nn1 + 1) * sizeof(double), hipMemcpyHostToDevice));

	///*	printf("%e ,%e ,%e ,%e\n", temp_host[4 * (nn1 + 1) + 4], fl.uu[4][4], temp_host[30 * (nn1 + 1) + 15], fl.uu[30][15]);
	//	PAUSE;*/
	//	HANDLE_ERROR(hipHostFree(temp_host));
	//}

	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGetLastError());

	//�������ճ��ϵ��
	double *Amu_dev;
	HANDLE_ERROR(hipMalloc((double **) & Amu_dev, (mm1+1)*(nn1+1)*sizeof(double)));
	//get_viscous_with_cuda << <blockPerGrid, threadPerBlock >> > (transferInt_dev, transferDouble_dev, Amu_dev, T);

	//��������ճ��ϵ��
	double * Amu_t_dev;
	HANDLE_ERROR(hipMalloc((double **)& Amu_t_dev, (mm1 + 1)*(nn1 + 1) * sizeof(double)));
	HANDLE_ERROR(hipMemset(Amu_t_dev, 0, (mm1 + 1)*(nn1 + 1) * sizeof(double)));

	//turbulence_SST_kw_before_cuda(Amu_t_dev, Amu_dev, d, uu, vv, T, U_dev, x_dev, y_dev, x1_dev, y1_dev, 
	//				mBlock, fl, transferInt_dev, transferDouble_dev); 

	HANDLE_ERROR(hipDeviceSynchronize());	//ͬ��
	HANDLE_ERROR(hipGetLastError());

	//����ͨ���� ͨ����С��������
	double * Fluxi_dev;	double * Fluxj_dev;
	HANDLE_ERROR(hipMalloc((double **)& Fluxi_dev, (nx+1)*(ny+1)*5 * sizeof(double)));
	HANDLE_ERROR(hipMemset(Fluxi_dev, 0, (nx + 1)*(ny + 1) * 5 * sizeof(double)));
	HANDLE_ERROR(hipMalloc((double **)& Fluxj_dev, (nx + 1)*(ny + 1) * 5 * sizeof(double)));
	HANDLE_ERROR(hipMemset(Fluxj_dev, 0, (nx + 1)*(ny + 1) * 5 * sizeof(double)));

	double * Fluxi_host;
	HANDLE_ERROR(hipHostAlloc((double **)&Fluxi_host, (nx + 1)*(ny + 1) * 5 * sizeof(double), hipHostMallocDefault));

	double * Fluxj_host;
	HANDLE_ERROR(hipHostAlloc((double **)&Fluxj_host, (nx + 1)*(ny + 1) * 5 * sizeof(double), hipHostMallocDefault));


	//ʹ�������������м���
	hipStream_t first_stream;
	HANDLE_ERROR(hipStreamCreate(&first_stream));
	hipStream_t second_stream;
	HANDLE_ERROR(hipStreamCreate(&second_stream));

	getResidual_i_WithKernel <<< blockPerGrid, threadPerBlock, 0, first_stream >>> ( transferInt_dev, transferDouble_dev,
								U_dev,Amu_dev, Amu_t_dev, x1_dev, y1_dev, x_dev, y_dev, Fluxi_dev, uu, vv, T);

	HANDLE_ERROR(hipMemcpyAsync(Fluxi_host, Fluxi_dev, (nx + 1)*(ny + 1) * 5 * sizeof(double),hipMemcpyDeviceToHost, first_stream));	//��ͨ��������������

	getResidual_j_WithKernel << < blockPerGrid, threadPerBlock,0, second_stream >> > (transferInt_dev, transferDouble_dev,
								U_dev, Amu_dev, Amu_t_dev, x1_dev, y1_dev, x_dev, y_dev, Fluxj_dev, uu, vv, T);

	HANDLE_ERROR(hipMemcpyAsync(Fluxj_host, Fluxj_dev, (nx + 1)*(ny + 1) * 5 * sizeof(double), hipMemcpyDeviceToHost, second_stream));			//��ͨ��������������

	HANDLE_ERROR(hipDeviceSynchronize());	//ͬ��
	HANDLE_ERROR(hipGetLastError());

	for (int i = 1; i <= nx-1; ++i) {
		for (int j = 1; j <= ny-1; ++j) {
			for (int k = 1; k <= 4; ++k) {
				int a = i*(ny + 1) * 5 + j * 5 + k;
				int b = (i + 1)*(ny + 1) * 5 + j * 5 + k;
				int c = i*(ny + 1) * 5 + (j + 1) * 5 + k;
				B.Res[i][j][k] =Fluxi_host[b]-Fluxi_host[a] + Fluxj_host[c] - Fluxj_host[a];

				//printf("B.Res[%d][%d][%d]=  %15.12e , %15.12e , %15.12e\n", i, j, k, B.Res[i][j][k], Fluxi_host[b], Fluxi_host[a]);

				if (B.Res[i][j][k] != B.Res[i][j][k] || abs(B.Res[i][j][k])>10) {
					//printf("@@B.Res[%d][%d][%d]=  %15.12e \n", i, j, k, B.Res[i][j][k]);
					//PAUSE;
				}
			}
			//PAUSE;
		}
	}
	HANDLE_ERROR(hipGetLastError());

	HANDLE_ERROR(hipHostFree(Fluxi_host));
	HANDLE_ERROR(hipHostFree(Fluxj_host));

	HANDLE_ERROR(hipFree(Fluxi_dev)); HANDLE_ERROR(hipFree(Fluxj_dev));
	HANDLE_ERROR(hipFree(uu));	 HANDLE_ERROR(hipFree(vv)); HANDLE_ERROR(hipFree(T));
	//HANDLE_ERROR(hipFree(x1_dev));	HANDLE_ERROR(hipFree(y1_dev));
	//HANDLE_ERROR(hipFree(x_dev));	HANDLE_ERROR(hipFree(y_dev));
	HANDLE_ERROR(hipFree(U_dev));
	HANDLE_ERROR(hipFree(Amu_dev));	HANDLE_ERROR(hipFree(Amu_t_dev));
	//HANDLE_ERROR(hipFree(transferInt_dev));	HANDLE_ERROR(hipFree(transferDouble_dev));
	
	HANDLE_ERROR(hipStreamDestroy(first_stream));
	HANDLE_ERROR(hipStreamDestroy(second_stream));
}

//���� i �������ֵͨ��
__global__
void getResidual_i_WithKernel(int * transferInt, double *transferDouble, double* U,double * Amu, double *Amu_t,
		double * x1, double *y1, double *x, double *y, double *tFluxi, double *uu, double *vv, double * T)
{
	__shared__  int  nx, ny, LAP, mm, nn, mm1, nn1;
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	nx = transferInt[0];	ny = transferInt[1];
	LAP = transferInt[2];
	mm = nx + 2 * LAP; nn = ny + 2 * LAP;
	mm1 = nx + 2 * LAP-1; nn1 = ny + 2 * LAP-1;
	__shared__ double p00, gamma, Cp, Pr, PrT;
	p00 = transferDouble[0];
	gamma = transferDouble[1];
	Cp = transferDouble[6];
	Pr = transferDouble[7];
	PrT = transferDouble[8];

	double Fluxi[5] = { 0 };		//����
	double UL[5], UR[5], UL2[5], UR2[5], QL[5], QR[5], Flux0[5];
	double U0[5][5];
	//$OMP ENDDO
	//---- - i - direction----------------------------------------------------------------------------------
	if (i >= 1 && i <= nx && j >= 1 && j <= ny - 1) {
			int flagL = (i + LAP)*(nn+1) + j + 1 + LAP;
			int flagR= (i + LAP)*(nn+1) + j + LAP;
			double dx = x[flagL] - x[flagR];
			double dy = y[flagL] - y[flagR];
			const double si = sqrt(dx*dx + dy*dy);	//�߳�
			const double ni1 = dy / si;
			const double ni2 = -dx / si;   //normal vector at(i, j) or (I - 1 / 2, J)
			int flag1 = (i + LAP)*(nn+1) + j+LAP;	int flag2 = (i + 1 + LAP)*(nn+1) + j + LAP;
			const double vol= abs((x[flag1] - x[flag2+1])*(y[flag2] - y[flag1+1]) -
				(x[flag2] - x[flag1+1])*(y[flag1] - y[flag2+1]))*0.5e0;

			for (int k = 1; k <= 4; ++k) {
				for (int m = 1; m <= 4; ++m) {
					int flag = (i - 3 + m + LAP)*(nn1 + 1) * 5 + (j + LAP) * 5 + k;
					U0[k][m] = U[flag];
				
				}
			}
			cuda_Reconstuction_Characteristic(U0, UL, UR, gamma);

			//------ - ������ת����ֱ�ڽ��� ������ - ���� ����ϵ��
			QL[1] = UL[1]; QL[2] = UL[2] * ni1 + UL[3] * ni2; QL[3] = -UL[2] * ni2 + UL[3] * ni1; QL[4] = UL[4]; //�ܶȡ�ѹ���������ٶȡ������ٶ� ����ֵ��
			QR[1] = UR[1]; QR[2] = UR[2] * ni1 + UR[3] * ni2; QR[3] = -UR[2] * ni2 + UR[3] * ni1; QR[4] = UR[4]; //�ܶȡ�ѹ���������ٶȡ������ٶ� ����ֵ��

			cuda_Flux_Van_Leer_1Da(QL, QR, Flux0, gamma);

			//------------------------------------------------
			//���ͨ�� ���任��x - y����ϵ��
			Fluxi[1] = -Flux0[1] * si;                            //����ͨ��
			Fluxi[2] = -(Flux0[2] * ni1 - Flux0[3] * ni2)*si;         //x - ������ͨ��
			Fluxi[3] = -(Flux0[2] * ni2 + Flux0[3] * ni1)*si;         //y - ������ͨ��
			Fluxi[4] = -Flux0[4] * si;                            //����ͨ��

		

			 //--i - ������ճͨ���������������i - ����ճ��ͨ��
			 //---------------------------------------------------------------------------------------------------------
			 //---------- - Viscous term--------------------------------------------------------------------------------
			 //��ɢϵ����ճ��ϵ�����ȴ���ϵ���� = ��������ֵ��ƽ�� ���߽紦���õ���ֵ)
			double Amu1 = 0; double Amk1 = 0;
			const int colum = nn1 + 1;
			if (i == 1) {
				const int flag = (i + LAP)*colum + j + LAP;
				Amu1 = Amu[flag] + Amu_t[flag];                  //ճ��ϵ��(���� + ����), �����ϵ�ֵ = ����ֵ��ƽ��
				Amk1 = Cp*(Amu[flag] / Pr + Amu_t[flag] / PrT);     //�ȴ���ϵ��
			}
			else if (i == nx) {
				const int flag = (i - 1 + LAP)*colum + j + LAP;
				Amu1 = Amu[flag] + Amu_t[flag];                  //ճ��ϵ��(���� + ����), �����ϵ�ֵ = ����ֵ��ƽ��
				Amk1 = Cp*(Amu[flag] / Pr + Amu_t[flag] / PrT);     //�ȴ���ϵ��
			}
			else {
				const int flag1 = (i - 1 + LAP)*colum + j + LAP;
				const int flag2 = (i + LAP)*colum + j + LAP;
				Amu1 = (Amu[flag1] + Amu[flag2] + Amu_t[flag1] + Amu_t[flag2])*0.5E0;                 //ճ��ϵ��(���� + ����), �����ϵ�ֵ = ����ֵ��ƽ��
				Amk1 = Cp*((Amu[flag1] + Amu[flag2]) / Pr + (Amu_t[flag1] + Amu_t[flag2]) / PrT)*0.5e0;   //�ȴ���ϵ��
			}

		/*	if (i == 1 && j == 1) {
				printf("Amu= %e, Amk= %e \n", Amu1, Amk1);
				printf("Cp= %e, Pr=%e,  PrT=%e \n", Cp, Pr, PrT);
				printf("Amu= %e, Amk= %e \n", Amu[flag], Amu_t[flag]);
			}*/

			//----Jocabianϵ�� ����������Լ�������ĵ���, ���ڼ����������ĵ�����
			int myi = i + LAP;	int myj = j + LAP;
			flag1 = myi*colum + myj;
			flag2 = (myi - 1)*colum + myj;
			const int flag3 = (myi - 1)*colum + myj + 1;
			const int flag4 = (myi - 1)*colum + myj - 1;
			double Dix =  x1[flag1] -  x1[flag2];
			double Diy =  y1[flag1] -  y1[flag2];
			double Djx = ( x1[flag3] +  x1[flag1 + 1] -  x1[flag4] -  x1[flag4])*0.25E0;
			double Djy = ( y1[flag3] +  y1[flag1 + 1] -  y1[flag4] -  y1[flag4])*0.25E0;
			double Ds = 1.E0 / (Dix*Djy - Djx*Diy);
			//�������Լ�������ĵ���
			double Diu =  uu[flag1] -  uu[flag2];
			double Div =  vv[flag1] -  vv[flag2];
			double DiT =  T[flag1] -  T[flag2];
			double Dju = ( uu[flag3] +  uu[flag1 + 1] -  uu[flag4] -  uu[flag4])*0.25E0;
			double Djv = ( vv[flag3] +  vv[flag1 + 1] -  vv[flag4] -  vv[flag4])*0.25E0;
			double DjT = ( T[flag3] +  T[flag1 + 1] -  T[flag4] -  T[flag4])*0.25E0;
			//��������x, y����ĵ���
			double ux = (Diu*Djy - Dju*Diy)*Ds;
			double vx = (Div*Djy - Djv*Diy)*Ds;
			double Tx = (DiT*Djy - DjT*Diy)*Ds;
			double uy = (-Diu*Djx + Dju*Dix)*Ds;
			double vy = (-Div*Djx + Djv*Dix)*Ds;
			double Ty = (-DiT*Djx + DjT*Dix)*Ds;
			//ճ��Ӧ��������ͨ��
			double u1 = ( uu[flag1] +  uu[flag2])*0.5E0;
			double v1 = ( vv[flag1] +  vv[flag2])*0.5E0;
			double t11 = ((4.E0 / 3.E0)*ux - (2.E0 / 3.E0)*vy)*Amu1;
			double t22 = ((4.E0 / 3.E0)*vy - (2.E0 / 3.E0)*ux)*Amu1;
			double t12 = (uy + vx)*Amu1;
			double E1 = u1*t11 + v1*t12 + Amk1*Tx;
			double E2 = u1*t12 + v1*t22 + Amk1*Ty;
			//���ճ��ͨ��
			Fluxi[2] += (t11*ni1 + t12*ni2)*si;
			Fluxi[3] += (t12*ni1 + t22*ni2)*si;
			Fluxi[4] += (E1*ni1 + E2*ni2)*si;

			int loc = i*(ny+1) * 5 + j * 5;
			for (int m = 1; m <= 4; ++m) {
				tFluxi[loc + m] = Fluxi[m];
			}

			//if (i == 1 && j == 1) {
			//	printf("\n debug i = %d, %d, %e , %e, %e, %e\n",i,j, Fluxi[1], Fluxi[2], Fluxi[3], Fluxi[4]);
			//	//printf("%10.9e, %10.9e, %10.9e, \n %10.9e, %10.9e, %10.9e, %10.9e, %10.9e\n ", si, ni1, ni2, t11, t12, t22, E1, E2);
			//	printf("%10.9e, %10.9e, %10.9e\n\n", put1, put2, put3);
			//}
			//if (i == 2 && j == 1) {
			//	printf("%d, %d, %e , %e, %e, %e\n", i, j, Fluxi[1], Fluxi[2], Fluxi[3], Fluxi[4]);
			//	printf("%10.9e, %10.9e, %10.9e\n\n", put1, put2, put3);
			//}
			//if (i == 1 && j == 2) {
			//	printf("%d, %d, %e , %e, %e, %e\n", i, j, Fluxi[1], Fluxi[2], Fluxi[3], Fluxi[4]);
			//	printf("%10.9e, %10.9e, %10.9e\n\n", put1, put2, put3);
			//}
	}
}


//���� j �������ֵͨ��
__global__
void getResidual_j_WithKernel(int * transferInt, double *transferDouble, double* U, double * Amu, double *Amu_t,
	double * x1, double *y1, double *x, double *y, double *tFluxj, double *uu, double *vv, double * T)
{
	__shared__  int  nx, ny, LAP, mm, nn, mm1, nn1, int colum;
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	nx = transferInt[0];	ny = transferInt[1];
	LAP = transferInt[2];
	mm = nx + 2 * LAP; nn = ny + 2 * LAP;
	mm1 = nx + 2 * LAP - 1; nn1 = ny + 2 * LAP - 1;
	colum = nn1 + 1;
	__shared__ double p00, gamma, Cp, Pr, PrT;
	p00 = transferDouble[0];
	gamma = transferDouble[1];
	Cp = transferDouble[6];
	Pr = transferDouble[7];
	PrT = transferDouble[8];

	double Fluxj[5] = { 0 };		//����
	double UL[5], UR[5], UL2[5], UR2[5], QL[5], QR[5], Flux0[5];
	double U0[5][5];

	//= == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == == =
	//j - �������ճ��ճ��ͨ��
	//---------------------------------------- - j - direction------------------------------------------------------------ -
	//$OMP DO
	if (i >= 1 && i <= nx - 1) {
		if (j >= 1 && j <= ny) {
			//�߳���������
			int flagL = (i + 1 + LAP)*(nn + 1) + j + LAP;
			int flagR = (i + LAP)*(nn + 1) + j + LAP;
			double dx = x[flagL] - x[flagR];
			double dy = y[flagL] - y[flagR];
			const double sj = sqrt(dx*dx + dy*dy);	//�߳�
			const double nj1 = -dy / sj;
			const double nj2 = dx / sj;   //normal vector at(i, j) or (I - 1 / 2, J)

			//�����غ�������ع�
			for (int k = 1; k <= 4; ++k) {
				for (int m = 1; m <= 4; ++m) {
					U0[k][m] = U[(i + LAP)*colum*5+(j - 3 + m + LAP)*5+k];
				}
			}
			cuda_Reconstuction_Characteristic(U0, UL, UR, gamma);
			
			//------ - ������ת����ֱ�ڽ��� ������ - ���� ����ϵ��
			QL[1] = UL[1]; QL[2] = UL[2] * nj1 + UL[3] * nj2; QL[3] = -UL[2] * nj2 + UL[3] * nj1; QL[4] = UL[4]; //�ܶȡ�ѹ���������ٶȡ������ٶ� ����ֵ��
			QR[1] = UR[1]; QR[2] = UR[2] * nj1 + UR[3] * nj2; QR[3] = -UR[2] * nj2 + UR[3] * nj1; QR[4] = UR[4]; //�ܶȡ�ѹ���������ٶȡ������ٶ� ����ֵ��

			cuda_Flux_Van_Leer_1Da(QL, QR, Flux0, gamma);

			//------------------------------------------------
			//���ͨ�� ���任��x - y����ϵ��
			Fluxj[1] = -Flux0[1] * sj;                            //����ͨ��
			Fluxj[2] = -(Flux0[2] * nj1 - Flux0[3] * nj2)*sj;         //x - ������ͨ��
			Fluxj[3] = -(Flux0[2] * nj2 + Flux0[3] * nj1)*sj;         //y - ������ͨ��
			Fluxj[4] = -Flux0[4] * sj;                            //����ͨ��

			double Amu1 = 0;	double Amk1 = 0;
			if (j == 1) {
				int flag = (i + LAP)*colum + j + LAP;
				Amu1 = Amu[flag] + Amu_t[flag];
				Amk1 = Cp*(Amu[flag] / Pr + Amu_t[flag] / PrT);   //�ȴ���ϵ��
			}
			else if (j == ny) {
				int flag = (i + LAP)*colum + j - 1 + LAP;
				Amu1 = Amu[flag] + Amu_t[flag];
				Amk1 = Cp*(Amu[flag] / Pr + Amu_t[flag] / PrT);   //�ȴ���ϵ��
			}
			else {
				int flag1 = (i + LAP)*colum + j + LAP;
				int flag2 = (i + LAP)*colum + j - 1 + LAP;
				Amu1 = (Amu[flag1] + Amu[flag2] + Amu_t[flag1] + Amu_t[flag2])*0.5E0;
				Amk1 = Cp*((Amu[flag2] + Amu[flag1]) / Pr + (Amu_t[flag2] + Amu_t[flag1]) / PrT)*0.5E0;   //�ȴ���ϵ��
			}

			int myi = i + LAP;	int myj = j + LAP;
			const int flag1 = (myi + 1)*colum + myj - 1;
			const int flag2 = (myi - 1)*colum + myj;
			const int flag3 = (myi - 1)*colum + myj - 1;
			const int flag4 = (myi + 1)*colum + myj;
			const int flag5 = myi*colum + myj;
			double Dix = ( x1[flag1] +  x1[flag4] -  x1[flag3] -  x1[flag2])*0.25E0;
			double Diy = ( y1[flag1] +  y1[flag4] -  y1[flag3] -  y1[flag2])*0.25E0;
			double Djx =  x1[flag5] -  x1[flag5 - 1];
			double Djy =  y1[flag5] -  y1[flag5 - 1];

			double Ds = 1.E0 / (Dix*Djy - Djx*Diy);

			double Diu = ( uu[flag1] +  uu[flag4] -  uu[flag3] -  uu[flag2])*0.25E0;
			double Div = ( vv[flag1] +  vv[flag4] -  vv[flag3] -  vv[flag2])*0.25E0;
			double DiT = ( T[flag1] +  T[flag4] -  T[flag3] -  T[flag2])*0.25E0;
			double Dju =  uu[flag5] -  uu[flag5 - 1];
			double Djv =  vv[flag5] -  vv[flag5 - 1];
			double DjT =  T[flag5] -  T[flag5 - 1];
			//
			double ux = (Diu*Djy - Dju*Diy)*Ds;
			double vx = (Div*Djy - Djv*Diy)*Ds;
			double Tx = (DiT*Djy - DjT*Diy)*Ds;
			double uy = (-Diu*Djx + Dju*Dix)*Ds;
			double vy = (-Div*Djx + Djv*Dix)*Ds;
			double Ty = (-DiT*Djx + DjT*Dix)*Ds;
			double t11 = ((4.E0 / 3.E0)*ux - (2.E0 / 3.E0)*vy)*Amu1;
			double t22 = ((4.E0 / 3.E0)*vy - (2.E0 / 3.E0)*ux)*Amu1;
			double t12 = (uy + vx)*Amu1;
			double	u1 = ( uu[flag5] +  uu[flag5 - 1])*0.5E0;
			double v1 = ( vv[flag5] +  vv[flag5 - 1])*0.5E0;

			double E1 = u1*t11 + v1*t12 + Amk1*Tx;
			double E2 = u1*t12 + v1*t22 + Amk1*Ty;

			Fluxj[2] += (t11*nj1 + t12*nj2)*sj;
			Fluxj[3] += (t12*nj1 + t22*nj2)*sj;
			Fluxj[4] += (E1*nj1 + E2*nj2)*sj;

	/*		if (i == 1 && j == 1) {
				printf(" debugj= %d, %d, %12.10e , %e, %e, %e\n", i, j, Fluxj[1], Fluxj[2], Fluxj[3], Fluxj[4]);
			}
			if (i == 16 && j == 29) {
				printf("%d, %d, %e , %e, %e, %e\n", i, j, Fluxj[1], Fluxj[2], Fluxj[3], Fluxj[4]);
			}
			if (i == 36 && j == 13) {
				printf(" %d, %d, %e , %e, %e, %e\n", i, j, Fluxj[1], Fluxj[2], Fluxj[3], Fluxj[4]);
			}*/

			int loc = i*(ny + 1) * 5 + j* 5;

			for (int m = 1; m <= 4; ++m) {
				tFluxj[loc + m] =Fluxj[m];
			}

			//printf("*&= %d,  %d, %10.9f, %10.9f, %10.9f, %10.9f\n", i, j, Fluxj[1], Fluxj[2], Fluxj[3], Fluxj[4]);

			//if (i == 14 && j == 6) {
			//	printf("*&= %d,  %d, %10.9f,   %f, %f, %f\n", nn1, mm1, Fluxj[1], Fluxj[2], Fluxj[3], Fluxj[4]);
			//	printf("Amu= %d,  %d,  %f, %f,  %f, %f \n", LAP, j, Res[locL + 1], Res[locL + 2], Res[locL + 3], Res[locL + 4]);
			//	printf("Amu= %d,  %d,  %f, %f,  %f, %f \n", LAP, j, Res[locR + 1], Res[locR + 2], Res[locR + 3], Res[locR + 4]);
			//	printf("Amu= %f,  %f,  %f, %f \n", U0[1][1], U0[2][2], U0[3][1], U0[4][3]);
			//	printf("Amu= %d,  %d,  %f, %f \n", i, j, nj1, nj2);
			//}

		}
	}

}


//----------------------------------------------------------
//�����غ���������������(d, u, v, T, p, c)
//----------------------------------------------------------
__global__
void comput_duvtpckw_with_cuda(int * transferInt, double *transferDouble, double *d,
			double * uu,double * vv, double *T,   double * U)
{
	__shared__  int nx, ny, LAP, mm1, nn1;
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	nx = transferInt[0];
	ny = transferInt[1];
	LAP = transferInt[2];
	__shared__ double p00, Cv, Ma;
	p00 = transferDouble[0];
	Cv = transferDouble[2];
	Ma = transferDouble[3];
	mm1 = nx + 2 * LAP - 1;
	nn1 = ny + 2 * LAP - 1;
	if (i >= 1 && i <= mm1) {
		if (j >= 1 && j <= nn1) {
			int flag = i*(nn1+1) + j;
			int flag2= i*(nn1 + 1)*5 + j*5;
			d[flag] = U[flag2 + 1];
			uu[flag] = U[flag2+2] / d[flag];
			vv[flag] = U[flag2+3] / d[flag];
			T[flag] = (U[flag2+4] - 0.5e0*d[flag] * (uu[flag] * uu[flag] + vv[flag] * vv[flag])) / (Cv*d[flag]);
			//if (i == 4 && j == 6) { printf("%d,  %d,  %f\n", i, j, d); }
		}
	}
}


//�������ճ��ϵ��
__global__
void get_viscous_with_cuda(int * transferInt, double *transferDouble, double *Amu, double * T)
{
	__shared__  int nx, ny, LAP, nxM, nyM, ny1;
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	nx = transferInt[0];
	ny = transferInt[1];
	LAP = transferInt[2];
	__shared__ double T_inf, Tsb, Re;
	T_inf = transferDouble[4];
	Tsb = 110.4E0 / T_inf;
	Re = transferDouble[5];

	nxM = nx + LAP;	nyM = ny + LAP;	ny1 = ny + 2 * LAP - 1;
	if (i >= LAP&&i <= nxM) {
		if (j >= LAP&&j <= nyM) {
			int flag = i*(ny1+1) + j;
			Amu[flag] = 1.E0 / Re*(1.E0 + Tsb)*sqrt(T[flag] * T[flag] * T[flag]) / (Tsb + T[flag]);
		}
	}
}


//�������������ع�
__device__
void cuda_Reconstuction_Characteristic(double U0[5][5], double *UL, double * UR, double gamma)
{
	//real * 8::Uh(4), S(4, 4), S1(4, 4), V0(4, 4), VL(4), VR(4), QL(4), QR(4)
	//real * 8::V2, d1, u1, v1, p1, c1, tmp0, tmp1, tmp3, tmp5

	//U0(m, k) : k = 1, 4 for  i - 2, i - 1, i, i + 1; m for the conservative variables U0(1, m) = d, U0(2, m) = d*u, ....
	double Uh[5] = { 0 };
	for (int i = 1; i <= 4; ++i) {
		Uh[i] = 0.5E0*(U0[i][2] + U0[i][3]);	  //conservative variables in the point I - 1 / 2  (or i)
	}
	double d1 = Uh[1]; double u1 = Uh[2] / d1; double v1 = Uh[3] / d1;
	double p1 = (Uh[4] - (Uh[2] * u1 + Uh[3] * v1)*0.5E0)*(gamma - 1.E0);  //density, velocity, pressure and sound speed
	double c1 = sqrt(gamma*p1 / d1);

	double V2 = (u1*u1 + v1*v1)*0.5E0;
	double tmp1 = (gamma - 1.E0) / c1;
	double tmp3 = (gamma - 1.E0) / (c1*c1);
	double tmp5 = 1.E0 / (2.E0*c1);
	double tmp0 = 1.E0 / tmp3;

	//A = S(-1)*LAMDA*S    see �������������ѧ�� 158 - 159ҳ(with alfa = 1, beta = 0)
	double S[5][5];		double S1[5][5];

	S[1][1] = V2 - tmp0;       S[1][2] = -u1;         S[1][3] = -v1;      S[1][4] = 1.E0;
	S[2][1] = -v1;          S[2][2] = 0.E0;        S[2][3] = 1.E0;     S[2][4] = 0.E0;
	S[3][1] = -u1 - V2*tmp1;   S[3][2] = 1.E0 + tmp1*u1; S[3][3] = tmp1*v1;   S[3][4] = -tmp1;
	S[4][1] = -u1 + V2*tmp1;   S[4][2] = 1.E0 - tmp1*u1; S[4][3] = -tmp1*v1;  S[4][4] = tmp1;

	S1[1][1] = -tmp3;    S1[1][2] = 0.E0;   S1[1][3] = -tmp5;         S1[1][4] = tmp5;
	S1[2][1] = -tmp3*u1; S1[2][2] = 0.E0;   S1[2][3] = 0.5E0 - u1*tmp5; S1[2][4] = 0.5E0 + u1*tmp5;
	S1[3][1] = -tmp3*v1; S1[3][2] = 1.E0;   S1[3][3] = -v1*tmp5;       S1[3][4] = v1*tmp5;
	S1[4][1] = -tmp3*V2; S1[4][2] = v1;     S1[4][3] = (c1*u1 - V2 - tmp0)* tmp5; S1[4][4] = (c1*u1 + V2 + tmp0) * tmp5;

	//V = SU      V[k] = S*U[k]
	double V0[5][5];
	for (int k = 1; k <= 4; ++k) {
		for (int m = 1; m <= 4; ++m) {
			V0[k][m] = 0.E0;
			for (int j = 1; j <= 4; ++j) {
				V0[k][m] = V0[k][m] + S[m][j] * U0[j][k];
			}
		}
	}
	double VL[5] = { 0 }; double VR[5] = { 0 };
	for (int m = 1; m <= 4; ++m) {
		cuda_scheme_fP(VL[m], V0[1][m], V0[2][m], V0[3][m], V0[4][m]);
		cuda_scheme_fm(VR[m], V0[1][m], V0[2][m], V0[3][m], V0[4][m]);
	}

	double QL[5] = { 0 }; double QR[5] = { 0 };
	for (int m = 1; m <= 4; ++m) {
		QL[m] = 0.E0; QR[m] = 0.E0;
		for (int j = 1; j <= 4; ++j) {
			QL[m] = QL[m] + S1[m][j] * VL[j];
			QR[m] = QR[m] + S1[m][j] * VR[j];
		}
	}

	UL[1] = QL[1]; UL[2] = QL[2] / UL[1]; UL[3] = QL[3] / UL[1];
	UL[4] = (QL[4] - (UL[2] * QL[2] + UL[3] * QL[3])*0.5E0)* (gamma - 1.E0);  //density, velocity, pressure and sound speed
	UR[1] = QR[1]; UR[2] = QR[2] / UR[1]; UR[3] = QR[3] / UR[1];
	UR[4] = (QR[4] - (UR[2] * QR[2] + UR[3] * QR[3]) * 0.5E0) * (gamma - 1.E0);  //find a bug, removed
}


//---------------------------------------------------------- -
//��ֵ��ʽ������UL = U(j + 1 / 2, L); u1 = u(j - 1), u2 = u(j), u3 = u(j + 1), u4 = u(j + 2)
__device__
void cuda_scheme_fP(double &UL, double u1, double u2, double u3, double u4)
{
	const double k = 1.E0;
	const double k3 = 1.E0 / 3.E0, ep = 1.E-6;

	//Iflag_Scheme == Scheme_MUSCL3         //3��MUSCL(Van Albada������)
	double up = u3 - u2; double um = u2 - u1;//1�� ǰ����
	double s = (2.E0*up*um + ep) / (up*up + um*um + ep);//Van Albada������ ���⻬����ǰ������ӽ�����ֵ�ӽ�1��
	UL = u2 + 0.25E0*s*((1.E0 - k3*s)*um + (1.E0 + k3*s)*up);//3��MUSCL(�⻬���ƽ�3��ӭ��)
}

//��ֵ��ʽ������UR = U(j + 1 / 2, R); u1 = u(j - 1), u2 = u(j), u3 = u(j + 1), u4 = u(j + 2)
__device__
void cuda_scheme_fm(double & UR, double u1, double u2, double u3, double u4)
{
	const double k = 1.E0 / 3.0;
	const double k3 = 1.E0 / 3.E0, ep = 1.E-6;
	
	//Iflag_Scheme == Scheme_MUSCL3      //3��MUSCL(Van Albada������)
	double up = u4 - u3; double um = u3 - u2;                             //ǰ����
	double s = (2.E0*up*um + ep) / (up*up + um*um + ep);
	UR = u3 - 0.25E0*s*((1.E0 - k3*s)*up + (1.E0 + k3*s)*um);
}


//---------------------------------------------------------- -
//Code by Cofludy according to Leng Yan`s code 
__device__
void  cuda_Flux_Van_Leer_1Da(double *QL, double * QR, double *Flux, const double gamma)
{
	double dl, uul, vvl, pl, al, dr, uur, vvr, pr, ar, Ml, Mr, Mp, Mm;  //uu velocity
	double tmp0, fp[5], fm[5];

	dl = QL[1]; uul = QL[2]; vvl = QL[3];  pl = QL[4];
	dr = QR[1]; uur = QR[2]; vvr = QR[3];  pr = QR[4];
	al = sqrt(gamma*pl / dl);  //density, velocity, pressure and sound speed
	ar = sqrt(gamma*pr / dr);
	Ml = uul / (al); Mr = uur / (ar);
	if (Ml >= 1.E0) {
		fp[1] = dl*uul;
		fp[2] = dl*uul*uul + pl;
		fp[3] = dl*uul*vvl;
		fp[4] = uul*(gamma*pl / (gamma - 1.E0) + 0.5E0*dl*(uul*uul + vvl*vvl));
	}
	else if (abs(Ml)<1.E0) {
		Mp = 0.25E0*(1.E0 + Ml)*(1.E0 + Ml);
		tmp0 = dl*al*Mp;
		fp[1] = tmp0;
		fp[2] = tmp0*((gamma - 1.E0)*uul + 2.E0*al) / gamma;
		fp[3] = tmp0*vvl;
		fp[4] = tmp0*(((gamma - 1.E0)*uul + 2.E0*al)*((gamma - 1.E0)*uul + 2.E0*al)*0.5E0 / (gamma*gamma - 1.E0) + 0.5E0*(vvl*vvl));
	}
	else if (Ml <= -1.E0) {
		fp[1] = 0.E0;
		fp[2] = 0.E0;
		fp[3] = 0.E0;
		fp[4] = 0.E0;
	}

	if (Mr >= 1.E0) {
		fm[1] = 0.E0;
		fm[2] = 0.E0;
		fm[3] = 0.E0;
		fm[4] = 0.E0;
	}
	else if (abs(Mr) < 1.E0) {
		Mm = -0.25E0*(Mr - 1.E0) * (Mr - 1.E0);
		tmp0 = dr*ar*Mm;
		fm[1] = tmp0;
		fm[2] = tmp0*((gamma - 1.E0) * uur - 2.E0*ar) / gamma;
		fm[3] = tmp0*vvr;
		fm[4] = tmp0*(((gamma - 1.E0)*uur - 2.E0*ar)*((gamma - 1.E0)*uur - 2.E0*ar)*0.5E0 / (gamma*gamma - 1.E0) + 0.5E0*(vvr*vvr));
	}
	else if (Mr <= -1.E0) {
		fm[1] = dr*uur;
		fm[2] = dr*uur*uur + pr;
		fm[3] = dr*uur*vvr;
		fm[4] = uur*(gamma*pr / (gamma - 1.E0) + 0.5E0*dr*(uur*uur + vvr*vvr));
	}
	for (int i = 1; i <= 4; ++i) {
		Flux[i] = fp[i] + fm[i];
	}

}