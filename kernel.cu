#include "hip/hip_runtime.h"
// this is the main function file 
//this peogram is a 2D fluid simulation using finite volume method with multiblocks structure mesh 
// code is accelerated by GPU device 
// allright keep by the writer cofludy 
// connect Cofludy@gmail.com


#include<fstream>
#include <string>
#include <iomanip>
#include<iostream>

#include "const_var.h"
#include "Global_var.h"
#include"sub_init.h"
#include "sub_boundary.h"
#include "sub_Finite_Difference.h"
#include "sub_turbulence_SST.h"
#include "common.h"
#include "sub_NS_singlegid.h"
#include "postAnalyze.h"


hipError_t CheckCudaDevice();
void read_parameter();
void set_control_para();
void output_Res(int nMesh);
void output(int nMesh);
void myFree();

int FileOpenFlag = 0;

extern bool USEGPU = true;

int main()
{
	printf("----------------- OpenCFD-EC2D in CUDA ver 2.0.0 --------------------------\n");
	printf("               Copyright by Lee Hiloong,  cofludy@gmail.com\n");
	printf("		  Programming by Lee HiLoong  2017-7-12        \n \n");
    // Add vectors in parallel.
    hipError_t cudaStatus = CheckCudaDevice();

	read_parameter();	//��ȡ����������������Ϣ

	check_mesh_multigrid();      //�������������������������, ���趨�������������
	Init();						// ��ʼ�����������ݽṹ; ���뼸�μ�������Ϣ
	set_control_para();			//�趨���������ϵĿ�����Ϣ����ֵ������ͨ������������ģ�͡�ʱ���ƽ���ʽ��
	Update_coordinate_buffer();		//����������Ϣ�����������������
	Init_FiniteDifference();		//�趨���޲�ַ������򣬼���Jocabian�任ϵ��

	if (Iflag_turbulence_model == Turbulence_SST || Iflag_turbulence_model == Turbulence_SA) {
		comput_dw();        //������������ģ��㵽����ľ��� ������SA��SSTģ��ʱ��Ҫ�˼���)
	}
	Init_flow();

	//Mesh_TYPE & MP = Mesh[1];
	//Block_TYPE & B = MP.Block[1];        //��nMesh ������ĵ�mBlock��
	//printf("%f,  %f,   %f,   %f", B.U[1][1][1], B.U[1][1][2], B.U[1][1][3], B.U[1][1][4]);
	//PAUSE;

	printf("start ... ...\n");

	//ͳ������ʱ��   
	clock_t start, finish;
	start = clock();


	//------------------------------------------------------------------------
	//ʱ���ƽ������õ������񡢶���������������� ����1��Euler, 3��RK��LU - SGS
	for (; Mesh[1].tt < t_end && Mesh[1].Kstep<2000; ) {
                //��������ʱ���ƽ�(1��Euler, 3��RK, LU - SGS)
			NS_Time_advance(1);

		if (Mesh[1].Kstep % Kstep_show == 0) {
			output_Res(1);         //��ӡ�в�(��������)
		}
		if (Mesh[1].Kstep % Kstep_save == 0) {
			output(1);           //�������(��������)
			/*outputPressureOnWall(1);*/
			outPutVelcoity();
		}

	}

	output(1);
	outPutVelcoity();


	finish = clock();
	double duration = (double)(finish - start) / CLOCKS_PER_SEC;
	std::cout << "running time = " << duration << " s  = " << duration / 60.0 << " min\n\n Press Any Button to Exit" << std::endl; 

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	PAUSE;
	myFree();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t CheckCudaDevice()
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(0));

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    return cudaStatus;
}




//��ȡ����������������Ϣ
void read_parameter()
{
	std::ifstream fcin;
	fcin.open("control.in");
	std::string tempStr;
	getline(fcin, tempStr);
	fcin >> Ma >> Re >> gamma >> AoA >> Pr >> t_end >> Kstep_save >> If_viscous >> Iflag_turbulence_model >> Iflag_init;
	getline(fcin, tempStr); getline(fcin, tempStr);
	fcin >> Iflag_local_dt >> dt_global >> CFL >> dtmax >> dtmin >> Time_Method >> p_outlet >> T_inf >> Twall >> vt_inf >> Kt_inf >> Wt_inf;
	getline(fcin, tempStr); getline(fcin, tempStr);
	fcin >> Iflag_Scheme >> Iflag_Flux >> IFlag_Reconstruction >> Kstep_show;
	getline(fcin, tempStr); getline(fcin, tempStr);
	fcin >> Num_Mesh >> Num_Threads >> Nstep_Inner_Limit >> Res_Inner_Limit;
	getline(fcin, tempStr); getline(fcin, tempStr);
	for (int i = 1; i <= Num_Mesh; ++i) {
		fcin >> Pre_Step_Mesh[i];
	}
	fcin.close();
	//printf("try= %d", Kstep_save);

	if ((Time_Method == Time_LU_SGS || Time_Method == Time_Dual_LU_SGS) && 1 != Num_Mesh) {
		printf("In this version (ver 1.5.1 ), LU_SGS method Do Not support Multigrid !!!\n");
		printf("Please modify 'control.in' to choose single-grid or other time method\n");
	}

	if (Iflag_turbulence_model == Turbulence_SST) {
		Nvar = 6;           //6������(4���������� + �Ķ���k + �Ⱥ�ɢ��w)
	}
	else if (Iflag_turbulence_model == Turbulence_SA) {
		Nvar = 5;
	}
	else {
		Nvar = 4;
	}

	AoA = AoA*PI / 180.e0;
	Cv = 1.e0 / (gamma*(gamma - 1.e0)*Ma*Ma);
	Cp = Cv*gamma;
	Twall = Twall / T_inf;
}

//�趨���������ϵĿ�����Ϣ
void set_control_para()
{
	Mesh_TYPE & MP = Mesh[1];	//��ϸ������
								//��ϸ������ϵĿ��Ʋ����������Ʋ�����ͬ
	MP.Iflag_turbulence_model = Iflag_turbulence_model;
	MP.Iflag_Scheme = Iflag_Scheme;
	MP.IFlag_flux = Iflag_Flux;
	MP.IFlag_Reconstruction = IFlag_Reconstruction;
	MP.Nvar = Nvar;		//���������̣���Ŀ����������ʹ������ģ�ͣ���Ŀ��Nvar��ͬ��

						//�趨�������ϵĿ��Ʋ���
	for (int nMesh = 2; nMesh <= Num_Mesh; ++nMesh) {
		Mesh_TYPE & MP = Mesh[nMesh];
		MP.Iflag_turbulence_model = Turbulence_NONE;
		MP.Iflag_Scheme = Scheme_UD1;
		MP.IFlag_flux = Iflag_Flux;
		MP.IFlag_Reconstruction = IFlag_Reconstruction;
		MP.Nvar = 4;	//���������̣���Ŀ����������ʹ������ģ�ͣ�������ĿΪ4��
	}
}


//-------------------------------------- -
//��ӡ�в���в�;������в
void output_Res(int nMesh)
{
	printf("\n\n  Kstep=  %d , t= %f  \n", Mesh[nMesh].Kstep, Mesh[nMesh].tt);
	printf("----------The Max Residuals are--------     ---Mesh---  %d\n", nMesh);
	for (int i = 1; i <= Nvar; ++i) {
		printf("%13.9f   ", Mesh[nMesh].Res_max[i]);
	}
	printf("\n");
	printf("  The R.M.S Residuals are \n");
	for (int i = 1; i <= Nvar; ++i) {
		printf("%13.9f   ", Mesh[nMesh].Res_rms[i]);
	}
	printf("\n");
	std::ofstream fcout;

	if (!FileOpenFlag) {
		FileOpenFlag = 1;
		fcout.open("Residual.dat");
		fcout.close();
	}

	fcout.open("Residual.dat", std::ios::app);
	fcout << std::setprecision(15);
	if (fcout.is_open()) {
		fcout << Mesh[nMesh].Kstep << "  ";
		for (int i = 1; i <= Nvar; ++i) {
			//printf("%13.9f   ", Mesh[nMesh].Res_max[i]);
			fcout << Mesh[nMesh].Res_max[i] << "  ";
		}
		//fcout << std::endl;
		for (int i = 1; i <= Nvar; ++i) {
			//printf("%13.9f   ", Mesh[nMesh].Res_rms[i]);
			fcout << Mesh[nMesh].Res_rms[i] << "  ";
		}
		fcout << std::endl;
		fcout.close();
	}
}


//----------------------------------------------------------------------
//������μ������� ��tecplot��ʽ��, ��ϸ����flow2d.dat; ������ flow2d - 2.dat; ������� flow2d - 3.dat
void output(int nMesh)
{
	std::string filename;
	if (nMesh == 1) {
		filename = "flow2d.dat";
	}
	else {
		filename = "flow2d-";
		std::string tempStr = std::to_string(nMesh);
		filename = filename + tempStr + ".plt";			// flow2d-2.dat ; flow2d-3.dat 
	}

	if (nMesh == 1) {
		std::string filename1 = "flow2d-";
		std::string tempStr1 = std::to_string(Mesh[1].Kstep);
		filename1 = filename1 + tempStr1 + ".plt";			// flow2d-2.dat ; flow2d-3.dat 
		Mesh_TYPE & MP = Mesh[1];
		std::ofstream fcout;
		fcout.open(filename1);
		fcout << std::setprecision(12);
		fcout << " variables=x,y,d,u,v,T,p " << std::endl;
		for (int m = 1; m <= MP.Num_Block; ++m) {
			Block_TYPE &B = Mesh[nMesh].Block[m];
			fcout << "zone  i= " << B.nx + 1 << "  j= " << B.ny + 1 << std::endl;
			for (int j = LAP; j <= B.ny + LAP; ++j) {
				for (int i = LAP; i <= B.nx + LAP; ++i) {
					double d1 = B.U[i][j][1];
					double u1 = B.U[i][j][2] / d1;
					double v1 = B.U[i][j][3] / d1;
					double T1 = (B.U[i][j][4] - 0.50*d1*(u1*u1 + v1*v1)) / (Cv*d1);
					double p1 = d1*T1 / (gamma*Ma*Ma);
					fcout << B.x1[i][j] << "  " << B.y1[i][j] << " " << d1 << " " << u1 << " " << v1 << " " << T1 << " " << p1 << std::endl;
				}
			}
		}
		fcout.close();
	}

	printf("write data file ...\n");
	Mesh_TYPE & MP = Mesh[nMesh];
	std::ofstream fcout;
	fcout.open(filename);
	fcout << std::setprecision(12);
	fcout << " variables=x,y,d,u,v,T,p,Amut " << std::endl;
	for (int m = 1; m <= MP.Num_Block; ++m) {
		Block_TYPE &B = Mesh[nMesh].Block[m];
		fcout << "zone  i= " << B.nx + 1 << "  j= " << B.ny + 1 << std::endl;
		for (int j = LAP; j <= B.ny + LAP; ++j) {
			for (int i = LAP; i <= B.nx + LAP; ++i) {
				double d1 = B.U[i][j][1];
				double u1 = B.U[i][j][2] / d1;
				double v1 = B.U[i][j][3] / d1;
				double T1 = (B.U[i][j][4] - 0.50*d1*(u1*u1 + v1*v1)) / (Cv*d1);
				fcout << B.x1[i][j] << "  " << B.y1[i][j] << " " << d1 << " " << u1 << " " << v1 << " " << T1 << " ";
				fcout << d1*T1 / (gamma*Ma*Ma) << "  " << B.Amu_t[i][j] * Re << std::endl;
			}
		}
	}
	fcout.close();

	if (MP.Nvar == 5) {
		fcout.open("SA2d.dat");
		fcout << std::setprecision(12);
		fcout << " variables=x,y,vt " << std::endl;
		for (int m = 1; m <= MP.Num_Block; ++m) {
			Block_TYPE &B = Mesh[nMesh].Block[m];
			fcout << "zone  i= " << B.nx + 1 << "  j= " << B.ny + 1 << std::endl;
			for (int j = LAP; j <= B.ny + LAP; ++j) {
				for (int i = LAP; i <= B.nx + LAP; ++i) {
					fcout << B.x1[i][j] << " " << B.y1[i][j] << " " << B.U[i][j][5] << std::endl;
				}
			}
		}
		fcout.close();
	}

	if (MP.Nvar == 6) {
		fcout.open("SST2D.dat");
		fcout << std::setprecision(12);
		fcout << " variables=x,y,Kt, Wt " << std::endl;
		for (int m = 1; m <= MP.Num_Block; ++m) {
			Block_TYPE &B = Mesh[nMesh].Block[m];
			fcout << "zone  i= " << B.nx + 1 << "  j= " << B.ny + 1 << std::endl;
			for (int j = LAP; j <= B.ny + LAP; ++j) {
				for (int i = LAP; i <= B.nx + LAP; ++i) {
					fcout << B.x1[i][j] << " " << B.y1[i][j] << " " << B.U[i][j][5] << " " << B.U[i][j][6] << std::endl;
				}
			}
		}
		fcout.close();
	}

}

void outputDebug()
{
#if debug
	Mesh_TYPE & MP = Mesh[1];
	std::ofstream fcout;
	fcout.open("outputDebug.plt");
	fcout << std::setprecision(12);
	fcout << " variables=x,y,d,u,v,T,p " << std::endl;
	for (int m = 1; m <= MP.Num_Block; ++m) {
		Block_TYPE &B = Mesh[1].Block[m];
		fcout << "zone  i= " << B.nx + 1 << "  j= " << B.ny + 1 << std::endl;
		for (int j = 1; j <= B.ny + 2 * LAP - 1; ++j) {
			for (int i = 1; i <= B.nx + 2 * LAP - 1; ++i) {
				double d1 = B.U[i][j][1];
				double u1 = B.U[i][j][2] / d1;
				double v1 = B.U[i][j][3] / d1;
				double T1 = (B.U[i][j][4] - 0.50*d1*(u1*u1 + v1*v1)) / (Cv*d1);
				double p1 = d1*T1 / (gamma*Ma*Ma);
				if (abs(d1) > 100 || abs(T1) > 100 || abs(u1) > 100 || abs(v1) > 100) {
					printf("%d, %d, \n %f, %f, %f, %f,", i, j, d1, u1, v1, T1);
					printf("this error is at %s file at %d  line", __FILE__, __LINE__);
					PAUSE;
				}
				fcout << B.x1[i][j] << "  " << B.y1[i][j] << " " << d1 << " " << u1 << " " << v1 << " " << T1 << " " << p1 << std::endl;
			}
		}
	}
	fcout.close();
#endif
}



void myFree()
{
	for (int iMesh = 1; iMesh <= Num_Mesh; ++iMesh) {
		Mesh_TYPE & MP = Mesh[iMesh];

		for (int iBlock = 1; iBlock <= MP.Num_Block; ++iBlock) {
			Block_TYPE & B = MP.Block[iBlock];
			int nx = B.nx;	int ny = B.ny;
			int mm = nx + 2 * LAP;	int nn = ny + 2 * LAP;
			deleteMatrix(B.x, mm);	deleteMatrix(B.y, mm);

			int mm1 = nx + 2 * LAP - 1;	int nn1 = ny + 2 * LAP - 1;
			deleteMatrix(B.x1, mm1);	deleteMatrix(B.y1, mm1);
			deleteMatrix(B.U, mm1, nn1);	deleteMatrix(B.deltU, mm1, nn1);
			deleteMatrix(B.Amu, mm1);	deleteMatrix(B.Amu_t, mm1);

			deleteMatrix(B.vol, nx);
			deleteMatrix(B.si, nx);		deleteMatrix(B.sj, nx);
			deleteMatrix(B.ni1, nx); deleteMatrix(B.ni2, nx);
			deleteMatrix(B.nj1, nx); deleteMatrix(B.nj2, nx);
			//�װ뾶
			deleteMatrix(B.Lci, nx); deleteMatrix(B.Lcj, nx);
			deleteMatrix(B.Lvi, nx); deleteMatrix(B.Lvj, nx);

			deleteMatrix(B.Un, nx, ny);	//��һʱ�䲽��ֵ
			if (Time_Method == Time_Dual_LU_SGS) {
				deleteMatrix(B.Un1, nx);		//n-1ʱ�䲽��ֵ�� ˫ʱ�䲽LU_SGS�����в���
			}
		}
	}
	free(Mesh);
	Mesh = NULL;
}
